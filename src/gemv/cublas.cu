
#include <benchmark/benchmark.h>

#include "gemv/args.hpp"
#include "init/init.hpp"
#include "utils/utils.hpp"

/* y = alpha * Ax + beta * y
hipblasStatus_t hipblasSgemv(hipblasHandle_t handle, hipblasOperation_t trans,
                           int m, int n,
                           const float           *alpha,
                           const float           *A, int lda,
                           const float           *x, int incx,
                           const float           *beta,
                           float           *y, int incy) */
static void CUBLAS_GEMV(benchmark::State &state) {

  const auto M_GLOBAL = state.range(0);
  const auto N_GLOBAL = state.range(1);

  const float alpha = 1.1f;
  const float beta  = 1.2f;

  float *a_fp32;
  float *x_fp32;
  float *y_fp32;

  PRINT_IF_ERROR(hipMalloc((void **) &a_fp32, M_GLOBAL * N_GLOBAL * sizeof(float)));
  PRINT_IF_ERROR(hipMalloc((void **) &x_fp32, N_GLOBAL * sizeof(float)));
  PRINT_IF_ERROR(hipMalloc((void **) &y_fp32, M_GLOBAL * sizeof(float)));

  hiprandGenerator_t gen;

  PRINT_IF_ERROR(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
  PRINT_IF_ERROR(hiprandSetPseudoRandomGeneratorSeed(gen, 1337ULL));
  PRINT_IF_ERROR(hiprandGenerateUniform(gen, a_fp32, M_GLOBAL * N_GLOBAL));
  PRINT_IF_ERROR(hiprandGenerateUniform(gen, x_fp32, N_GLOBAL));
  PRINT_IF_ERROR(hiprandGenerateUniform(gen, y_fp32, M_GLOBAL));

  PRINT_IF_ERROR(hiprandDestroyGenerator(gen));

  hipEvent_t start, stop;
  PRINT_IF_ERROR(hipEventCreate(&start));
  PRINT_IF_ERROR(hipEventCreate(&stop));

  hipblasHandle_t cublasHandle;
  PRINT_IF_ERROR(hipblasCreate(&cublasHandle));
  // Not use tensor cores
  PRINT_IF_ERROR(hipblasSetMathMode(cublasHandle, HIPBLAS_DEFAULT_MATH));

  const int incx = 1;
  const int incy = 1;

  for (auto _ : state) {
    PRINT_IF_ERROR(hipEventRecord(start));

    PRINT_IF_ERROR(hipblasSgemv(cublasHandle, HIPBLAS_OP_N, M_GLOBAL, N_GLOBAL, &alpha,
                               a_fp32, M_GLOBAL, x_fp32, incx, &beta, y_fp32, incy));

    PRINT_IF_ERROR(hipEventRecord(stop));
    PRINT_IF_ERROR(hipEventSynchronize(stop));

    state.PauseTiming();

    float msecTotal = 0.0f;
    PRINT_IF_ERROR(hipEventElapsedTime(&msecTotal, start, stop));
    state.SetIterationTime(msecTotal / 1000);
    state.ResumeTiming();
  }

  hipEventDestroy(start);
  hipEventDestroy(stop);

  PRINT_IF_ERROR(hipFree(a_fp32));
  PRINT_IF_ERROR(hipFree(x_fp32));
  PRINT_IF_ERROR(hipFree(y_fp32));

  hipDeviceReset();

  state.counters.insert({{"M", M_GLOBAL},
                         {"N", N_GLOBAL},
                         {"num_elements", M_GLOBAL * N_GLOBAL},
                         {"flops",
                          {state.iterations() * 2.0 * M_GLOBAL * N_GLOBAL,
                           benchmark::Counter::kAvgThreadsRate}}});
}

BENCHMARK(CUBLAS_GEMV)->ARGS()->UseManualTime();
