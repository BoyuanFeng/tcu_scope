#include "hip/hip_runtime.h"

#include <benchmark/benchmark.h>

#include "gemv/args.hpp"
#include "init/init.hpp"
#include "utils/utils.hpp"

#include <mma.h>
using namespace nvcuda;

#ifndef WARP_SIZE
#define WARP_SIZE (32)
#endif // WARP_SIZE

// MMA matrix tile dimensions. (16, 16, 16), (32, 8, 16), and (8, 32, 16) are
// currently supported.
static const int M = 16;
static const int N = 16;
static const int K = 16;

// Implementation constants.
// number of warps needed for col and row in one block
static const int BLOCK_COL_WARPS = 1;
static const int BLOCK_ROW_WARPS = 9;

// number of WMMA tiles (16 X 16) processed by one warp
static const int WARP_COL_TILES = 1;
static const int WARP_ROW_TILES = 1;

// number of WMMA tiles for col and rwo in one block
static const int BLOCK_COL_TILES = WARP_COL_TILES * BLOCK_COL_WARPS;
static const int BLOCK_ROW_TILES = WARP_ROW_TILES * BLOCK_ROW_WARPS;

// number of warps and threads in one block
static const int WARPS_PER_BLOCK   = BLOCK_ROW_WARPS * BLOCK_COL_WARPS;
static const int THREADS_PER_BLOCK = WARP_SIZE * WARPS_PER_BLOCK;

// each block processes one tile at a time
static const int TILE_WIDTH_M = BLOCK_ROW_TILES * M;
static const int TILE_WIDTH_N = BLOCK_COL_TILES * N; // TILE_WIDTH_N <= TILE_WIDTH_M
static const int TILE_WIDTH_K = TILE_WIDTH_M;        // TILE_WIDTH_K <= TILE_WIDTH_M

static __global__ void compute_wmma_gemv_sharedmem(const half *__restrict__ a,
                                                   const half *__restrict__ b, float *c,
                                                   int M_GLOBAL, int N_GLOBAL,
                                                   int K_GLOBAL, float alpha,
                                                   float beta) {

  __shared__ half subTileA[TILE_WIDTH_K][TILE_WIDTH_M];
  __shared__ half subTileB[TILE_WIDTH_N][TILE_WIDTH_K];

  int tx = threadIdx.x;
  // int ty = 0; // threadIdx.y;
  int tid = tx; // threadIdx.y * blockDim.x + threadIdx.x; // thread id in the block

  int aRow = blockIdx.x * TILE_WIDTH_M; // staring row of the current block in matrix A
  // int bCol = 0; // blockIdx.y * TILE_WIDTH_N; // staring col of the current
  // block in matrix B

  // Declare the fragments
  wmma::fragment<wmma::matrix_a, M, N, K, half, wmma::col_major> a_frag;
  wmma::fragment<wmma::matrix_b, M, N, K, half, wmma::col_major> b_frag;
  wmma::fragment<wmma::accumulator, M, N, K, float> acc_frag;
  wmma::fragment<wmma::accumulator, M, N, K, float> c_frag;
  wmma::fill_fragment(acc_frag, 0.0f);

  for (int k = 0; k < K_GLOBAL; k += TILE_WIDTH_K) {
    // Collaborative loading of M tiles into shared memory
    for (int i = 0; i < TILE_WIDTH_M * TILE_WIDTH_K; i += THREADS_PER_BLOCK) {
      int idx = (tid + i);
      int aX  = idx % TILE_WIDTH_M;
      int aY  = idx / TILE_WIDTH_M;

      if (((k + aY) < K_GLOBAL) && ((aRow + aX) < M_GLOBAL)) {
        subTileA[aY][aX] = a[(k + aY) * M_GLOBAL + aRow + aX];
      } else {
        subTileA[aY][aX] = half(0);
      }
    }

    // Collaborative loading N tiles into shared memory
    for (int i = 0; i < TILE_WIDTH_K * TILE_WIDTH_N; i += THREADS_PER_BLOCK) {
      int idx = (tid + i);
      int bX  = idx % TILE_WIDTH_K;
      int bY  = idx / TILE_WIDTH_K;

      if ((bY < N_GLOBAL) && ((k + bX) < K_GLOBAL)) {
        subTileB[bY][bX] = b[bY * K_GLOBAL + k + bX];
        // subTileB[bY][bX] = (((bCol + bY) < N_GLOBAL) && ((k + bX) <
        // K_GLOBAL)) ? b[(bCol + bY) * K_GLOBAL + k + bX] : half(0);
      } else {
        subTileB[bY][bX] = half(0);
      }
    }

    __syncthreads();

    for (int i = 0; i < TILE_WIDTH_K; i += K) {
      int subtileARow = M * (threadIdx.x / WARP_SIZE);
      int subtileACol = i;

      int subtileBRow = i;
      // int subtileBCol = 0; // N * threadIdx.y;

      // Load the inputs
      wmma::load_matrix_sync(a_frag,
                             (half *) subTileA + subtileARow + subtileACol * TILE_WIDTH_M,
                             TILE_WIDTH_M);
      wmma::load_matrix_sync(b_frag, (half *) subTileB + subtileBRow, TILE_WIDTH_K);

      // Perform the matrix multiplication
      wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
    }
  }

  // Load in the current value of c, scale it by beta, and add this our result
  // scaled by alpha
  int warpM = (blockIdx.x * blockDim.x + tx) / WARP_SIZE;
  // int warpN = 0; // blockIdx.y * blockDim.y + ty;
  int cRow = warpM * M;
  // int cCol  = 0; // warpN * N;

  if (cRow < M_GLOBAL) {
    wmma::load_matrix_sync(
        c_frag, c + cRow, M_GLOBAL,
        wmma::mem_col_major); // wmma::load_matrix_sync(c_frag, c + cRow + cCol
                              // * K_GLOBAL, M_GLOBAL, wmma::mem_col_major);

    for (int i = 0; i < c_frag.num_elements; i++) {
      c_frag.x[i] = alpha * acc_frag.x[i] + beta * c_frag.x[i];
    }
    // Store the output
    wmma::store_matrix_sync(c + cRow, c_frag, K_GLOBAL,
                            wmma::mem_col_major); // wmma::store_matrix_sync(c + cRow +
                                                  // cCol * K_GLOBAL, c_frag, K_GLOBAL,
                                                  // wmma::mem_col_major);
  }
}

static void CUDA_WMMA_GEMV_SHAREDMEM(benchmark::State &state) {
  const auto M_GLOBAL = state.range(0);
  const auto K_GLOBAL = state.range(1);
  const auto N_GLOBAL = BLOCK_COL_TILES * N;

  const float alpha = 1.1f;
  const float beta  = 1.2f;

  float *a_fp32;
  float *x_fp32;
  float *y;

  half *a_fp16;
  half *x_fp16;
  half *b_fp16;

  PRINT_IF_ERROR(hipMalloc((void **) &a_fp32, M_GLOBAL * K_GLOBAL * sizeof(float)));
  PRINT_IF_ERROR(hipMalloc((void **) &x_fp32, K_GLOBAL * sizeof(float)));
  PRINT_IF_ERROR(hipMalloc((void **) &y,
                            M_GLOBAL * N_GLOBAL *
                                sizeof(float))); // the first column holds the result

  PRINT_IF_ERROR(hipMalloc((void **) &a_fp16, M_GLOBAL * K_GLOBAL * sizeof(half)));
  PRINT_IF_ERROR(hipMalloc((void **) &x_fp16, K_GLOBAL * sizeof(half)));
  PRINT_IF_ERROR(hipMalloc((void **) &b_fp16, K_GLOBAL * N_GLOBAL * sizeof(half)));

  hiprandGenerator_t gen;

  PRINT_IF_ERROR(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
  PRINT_IF_ERROR(hiprandSetPseudoRandomGeneratorSeed(gen, 1337ULL));
  PRINT_IF_ERROR(hiprandGenerateUniform(gen, a_fp32, M_GLOBAL * N_GLOBAL));
  PRINT_IF_ERROR(hiprandGenerateUniform(gen, x_fp32, N_GLOBAL));
  PRINT_IF_ERROR(hiprandGenerateUniform(gen, y, M_GLOBAL));

  PRINT_IF_ERROR(hiprandDestroyGenerator(gen));

  // hiprand doesn't currently support fp16 so we generate in fp32 and convert to
  // fp16.
  PRINT_IF_LAUNCH_ERROR((convertFp32ToFp16<<<(M_GLOBAL * K_GLOBAL + 255) / 256, 256>>>(
      a_fp16, a_fp32, M_GLOBAL * K_GLOBAL)));
  PRINT_IF_LAUNCH_ERROR(
      (convertFp32ToFp16<<<(N_GLOBAL + 255) / 256, 256>>>(x_fp16, x_fp32, K_GLOBAL)));

  // copy vector x to matrix b, column-major
  PRINT_IF_ERROR(
      hipMemcpy(b_fp16, x_fp16, K_GLOBAL * sizeof(half), hipMemcpyDeviceToDevice));

  hipEvent_t start, stop;
  PRINT_IF_ERROR(hipEventCreate(&start));
  PRINT_IF_ERROR(hipEventCreate(&stop));

  dim3 gridDim;
  dim3 blockDim;

  blockDim.x = BLOCK_ROW_TILES * WARP_SIZE;
  blockDim.y = BLOCK_COL_TILES;

  gridDim.x = (M_GLOBAL + (M * BLOCK_ROW_TILES - 1)) / (M * BLOCK_ROW_TILES);
  gridDim.y = (N_GLOBAL + N * BLOCK_COL_TILES - 1) / (N * BLOCK_COL_TILES); // 1

  for (auto _ : state) {
    PRINT_IF_ERROR(hipEventRecord(start));

    (compute_wmma_gemv_sharedmem<<<gridDim, blockDim>>>(a_fp16, b_fp16, y, M_GLOBAL,
                                                        N_GLOBAL, K_GLOBAL, alpha, beta));

    PRINT_IF_ERROR(hipEventRecord(stop));
    PRINT_IF_ERROR(hipEventSynchronize(stop));

    state.PauseTiming();

    float msecTotal = 0.0f;
    PRINT_IF_ERROR(hipEventElapsedTime(&msecTotal, start, stop));
    state.SetIterationTime(msecTotal / 1000);
    state.ResumeTiming();
  }

  hipEventDestroy(start);
  hipEventDestroy(stop);

  PRINT_IF_ERROR(hipFree(a_fp32));
  PRINT_IF_ERROR(hipFree(x_fp32));
  PRINT_IF_ERROR(hipFree(y));
  PRINT_IF_ERROR(hipFree(a_fp16));
  PRINT_IF_ERROR(hipFree(x_fp16));
  PRINT_IF_ERROR(hipFree(b_fp16));

  hipDeviceReset();

  state.counters.insert({{"M", M_GLOBAL},
                         {"N", K_GLOBAL},
                         {"num_elements", M_GLOBAL * K_GLOBAL},
                         {"flops",
                          {state.iterations() * 2.0 * M_GLOBAL * K_GLOBAL,
                           benchmark::Counter::kAvgThreadsRate}}});
}

// BENCHMARK(CUDA_WMMA_GEMV_SHAREDMEM)->ARGS()->UseManualTime();
