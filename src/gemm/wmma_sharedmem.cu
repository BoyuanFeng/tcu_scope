#include "hip/hip_runtime.h"

#include <benchmark/benchmark.h>

#include "gemm/args.hpp"
#include "init/init.hpp"
#include "utils/utils.hpp"

#include <mma.h>
using namespace nvcuda;

#ifndef WARP_SIZE
#define WARP_SIZE (32)
#endif // WARP_SIZE

// MMA matrix tile dimensions. (16, 16, 16), (32, 8, 16), and (8, 32, 16) are
// currently supported.
static const int M = 16;
static const int N = 16;
static const int K = 16;

// Implementation constants.
// number of warps needed for col and row in one block
static const int BLOCK_COL_WARPS = 4;
static const int BLOCK_ROW_WARPS = 4;

// number of WMMA tiles (16 X 16) processed by one warp
static const int WARP_COL_TILES = 1;
static const int WARP_ROW_TILES = 1;

// number of WMMA tiles for col and rwo in one block
static const int BLOCK_COL_TILES = WARP_COL_TILES * BLOCK_COL_WARPS;
static const int BLOCK_ROW_TILES = WARP_ROW_TILES * BLOCK_ROW_WARPS;

// number of warps and threads in one block
static const int WARPS_PER_BLOCK   = BLOCK_ROW_WARPS * BLOCK_COL_WARPS;
static const int THREADS_PER_BLOCK = WARP_SIZE * WARPS_PER_BLOCK;

// each block processes one tile at a time
static const int TILE_WIDTH_M = BLOCK_ROW_TILES * M;
static const int TILE_WIDTH_N = BLOCK_COL_TILES * N; // TILE_WIDTH_N <= TILE_WIDTH_M
static const int TILE_WIDTH_K = TILE_WIDTH_M;        // TILE_WIDTH_K <= TILE_WIDTH_M

static __global__ void compute_gemm_sharedmem(const half *__restrict__ a,
                                              const half *__restrict__ b, float *c,
                                              int M_GLOBAL, int N_GLOBAL, int K_GLOBAL,
                                              float alpha, float beta) {

  __shared__ half subTileA[TILE_WIDTH_K][TILE_WIDTH_M];
  __shared__ half subTileB[TILE_WIDTH_N][TILE_WIDTH_K];

  int tx  = threadIdx.x;
  int ty  = threadIdx.y;
  int tid = threadIdx.y * blockDim.x + threadIdx.x; // thread id in the block

  int aRow = blockIdx.x * TILE_WIDTH_M; // staring row of the current block in matrix A
  int bCol = blockIdx.y * TILE_WIDTH_N; // staring col of the current block in matrix B

  // Declare the fragments
  wmma::fragment<wmma::matrix_a, M, N, K, half, wmma::col_major> a_frag;
  wmma::fragment<wmma::matrix_b, M, N, K, half, wmma::col_major> b_frag;
  wmma::fragment<wmma::accumulator, M, N, K, float> acc_frag;
  wmma::fragment<wmma::accumulator, M, N, K, float> c_frag;
  wmma::fill_fragment(acc_frag, 0.0f);

  for (int k = 0; k < K_GLOBAL; k += TILE_WIDTH_K) {
    // Collaborative loading of M and N tiles into shared memory
    for (int i = 0; i < TILE_WIDTH_M * TILE_WIDTH_K; i += THREADS_PER_BLOCK) {
      int idx          = (tid + i);
      int aX           = idx % TILE_WIDTH_M;
      int aY           = idx / TILE_WIDTH_M;
      int bX           = idx % TILE_WIDTH_K;
      int bY           = idx / TILE_WIDTH_K;
      subTileA[aY][aX] = (((k + aY) < K_GLOBAL) && ((aRow + aX) < M_GLOBAL))
                             ? a[(k + aY) * M_GLOBAL + aRow + aX]
                             : half(0);
      subTileB[bY][bX] = (((bCol + bY) < N_GLOBAL) && ((k + bX) < K_GLOBAL))
                             ? b[(bCol + bY) * K_GLOBAL + k + bX]
                             : half(0);
      //  printf("k=%d, aX=%d, aY=%d, bX=%d, bY=%d is and sm=%f and sn=%f \n",
      //  k, aX, aY, bX, bY, (float) subTileA[aY][aX], (float)
      //  subTileB[bY][bX]);
    }
    __syncthreads();

    for (int i = 0; i < TILE_WIDTH_K; i += K) {
      int subtileARow = M * (threadIdx.x / WARP_SIZE);
      int subtileACol = i;

      int subtileBRow = i;
      int subtileBCol = N * threadIdx.y;

      // Load the inputs
      wmma::load_matrix_sync(a_frag,
                             (half *) subTileA + subtileARow + subtileACol * TILE_WIDTH_M,
                             TILE_WIDTH_M);
      wmma::load_matrix_sync(b_frag,
                             (half *) subTileB + subtileBRow + subtileBCol * TILE_WIDTH_K,
                             TILE_WIDTH_K);

      // Perform the matrix multiplication
      wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
    }
  }

  // Load in the current value of c, scale it by beta, and add this our result
  // scaled by alpha
  int warpM = (blockIdx.x * blockDim.x + tx) / WARP_SIZE;
  int warpN = blockIdx.y * blockDim.y + ty;
  int cRow  = warpM * M;
  int cCol  = warpN * N;

  if (cRow < M_GLOBAL && cCol < N_GLOBAL) {
    wmma::load_matrix_sync(c_frag, c + cRow + cCol * K_GLOBAL, M_GLOBAL,
                           wmma::mem_col_major);

    for (int i = 0; i < c_frag.num_elements; i++) {
      c_frag.x[i] = alpha * acc_frag.x[i] + beta * c_frag.x[i];
    }
    // Store the output
    wmma::store_matrix_sync(c + cRow + cCol * K_GLOBAL, c_frag, K_GLOBAL,
                            wmma::mem_col_major);
  }
}

static void CUDA_WMMA_GEMM_SHAREDMEM(benchmark::State &state) {
  /* if (!has_cuda) { */
  /*   state.SkipWithError(fmt::format("CUDA_WMMA_GEMM_NAIVE no CUDA device
   * found")); */
  /*   return; */
  /* } */

  // M_GLOBAL, N_GLOBAL, K_GLOBAL must be multiple of M, N and K
  const auto M_GLOBAL = state.range(0);
  const auto N_GLOBAL = state.range(1);
  const auto K_GLOBAL = state.range(2);

  const float alpha = 1.0f;
  const float beta  = 1.0f;

  float *a_fp32;
  float *b_fp32;
  float *c;
  half *a_fp16;
  half *b_fp16;

  hiprandGenerator_t gen;

  // Use tensor cores
  PRINT_IF_ERROR(hipMalloc((void **) &a_fp32, M_GLOBAL * K_GLOBAL * sizeof(float)));
  PRINT_IF_ERROR(hipMalloc((void **) &b_fp32, K_GLOBAL * N_GLOBAL * sizeof(float)));
  PRINT_IF_ERROR(hipMalloc((void **) &a_fp16, M_GLOBAL * K_GLOBAL * sizeof(half)));
  PRINT_IF_ERROR(hipMalloc((void **) &b_fp16, K_GLOBAL * N_GLOBAL * sizeof(half)));

  PRINT_IF_ERROR(hipMalloc((void **) &c, M_GLOBAL * N_GLOBAL * sizeof(float)));

  PRINT_IF_ERROR(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
  PRINT_IF_ERROR(hiprandSetPseudoRandomGeneratorSeed(gen, 1337ULL));

  PRINT_IF_ERROR(hiprandGenerateUniform(gen, a_fp32, M_GLOBAL * K_GLOBAL));
  PRINT_IF_ERROR(hiprandGenerateUniform(gen, b_fp32, K_GLOBAL * N_GLOBAL));

  // hiprand doesn't currently support fp16 so we generate in fp32 and convert to
  // fp16.
  PRINT_IF_LAUNCH_ERROR((convertFp32ToFp16<<<(M_GLOBAL * K_GLOBAL + 255) / 256, 256>>>(
      a_fp16, a_fp32, M_GLOBAL * K_GLOBAL)));
  PRINT_IF_LAUNCH_ERROR((convertFp32ToFp16<<<(K_GLOBAL * N_GLOBAL + 255) / 256, 256>>>(
      b_fp16, b_fp32, K_GLOBAL * N_GLOBAL)));

  PRINT_IF_ERROR(hiprandGenerateUniform(gen, c, M_GLOBAL * N_GLOBAL));
  PRINT_IF_ERROR(hiprandDestroyGenerator(gen));

  hipEvent_t start, stop;
  PRINT_IF_ERROR(hipEventCreate(&start));
  PRINT_IF_ERROR(hipEventCreate(&stop));

  dim3 gridDim;
  dim3 blockDim;

  blockDim.x = BLOCK_ROW_TILES * WARP_SIZE;
  blockDim.y = BLOCK_COL_TILES;

  gridDim.x = (M_GLOBAL + (TILE_WIDTH_M - 1)) / TILE_WIDTH_M;
  gridDim.y = (N_GLOBAL + (TILE_WIDTH_N - 1)) / TILE_WIDTH_N;

  PRINT_IF_ERROR(hipDeviceSynchronize());

  for (auto _ : state) {
    PRINT_IF_ERROR(hipEventRecord(start));

    (compute_gemm_sharedmem<<<gridDim, blockDim>>>(a_fp16, b_fp16, c, M_GLOBAL, N_GLOBAL,
                                                   K_GLOBAL, alpha, beta));

    PRINT_IF_ERROR(hipEventRecord(stop));
    PRINT_IF_ERROR(hipEventSynchronize(stop));

    state.PauseTiming();

    float msecTotal = 0.0f;
    PRINT_IF_ERROR(hipEventElapsedTime(&msecTotal, start, stop));
    state.SetIterationTime(msecTotal / 1000);
    state.ResumeTiming();
  }

  state.counters.insert({{"M", M_GLOBAL},
                         {"N", N_GLOBAL},
                         {"K", K_GLOBAL},
                         {"num_elements", M * N * K},
                         {"flops",
                          {state.iterations() * 2.0 * M_GLOBAL * N_GLOBAL * K_GLOBAL,
                           benchmark::Counter::kAvgThreadsRate}}});

  hipEventDestroy(start);
  hipEventDestroy(stop);

  PRINT_IF_ERROR(hipFree(a_fp32));
  PRINT_IF_ERROR(hipFree(b_fp32));
  PRINT_IF_ERROR(hipFree(a_fp16));
  PRINT_IF_ERROR(hipFree(b_fp16));
  PRINT_IF_ERROR(hipFree(c));

  hipDeviceReset();
}

// BENCHMARK(CUDA_WMMA_GEMM_SHAREDMEM)->ARGS()->UseManualTime();
