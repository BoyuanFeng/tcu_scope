#include "hip/hip_runtime.h"
#define CUB_HALF_OPTIMIZATION 1

#include <benchmark/benchmark.h>

#include "init/init.hpp"
#include "reduction/args.hpp"
#include "utils/utils.hpp"

#include <hipcub/hipcub.hpp>

using namespace hipcub;

static void CUB_FULL_REDUCTION(benchmark::State &state) {
  const size_t num_elements = state.range(0);

  half *d_in_fp16 = nullptr;
  half *d_out     = nullptr;
  hipEvent_t start, stop;

  // Request and allocate temporary storage
  void *d_temp_storage      = NULL;
  size_t temp_storage_bytes = 0;

  try {
    PRINT_IF_ERROR(hipMalloc(&d_in_fp16, num_elements * sizeof(half)));
    PRINT_IF_ERROR(hipMalloc(&d_out, 1 * sizeof(half)));

    cuda_memory_set(d_in_fp16, 0.001f, num_elements);

    PRINT_IF_ERROR(hipDeviceSynchronize());

    PRINT_IF_ERROR(DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in_fp16, d_out,
                                     num_elements));

    PRINT_IF_ERROR(hipMalloc(&d_temp_storage, temp_storage_bytes));

    PRINT_IF_ERROR(hipEventCreate(&start));
    PRINT_IF_ERROR(hipEventCreate(&stop));

    defer(hipEventDestroy(start));
    defer(hipEventDestroy(stop));

    for (auto _ : state) {
      PRINT_IF_ERROR(hipEventRecord(start));

      PRINT_IF_ERROR(DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in_fp16,
                                       d_out, num_elements));

      PRINT_IF_ERROR(hipEventRecord(stop));
      PRINT_IF_ERROR(hipEventSynchronize(stop));

      state.PauseTiming();

      float msecTotal = 0.0f;
      PRINT_IF_ERROR(hipEventElapsedTime(&msecTotal, start, stop));
      state.SetIterationTime(msecTotal / 1000);
      state.ResumeTiming();
    }

    state.counters.insert({{"num_elements", num_elements},
                           {"flops",
                            {state.iterations() * 1.0 * num_elements,
                             benchmark::Counter::kAvgThreadsRate}}});

#if 0
  half h_out;
  PRINT_IF_ERROR(
      hipMemcpy(&h_out, d_out, 1 * sizeof(half), hipMemcpyDeviceToHost));

  int errors        = 0;
  float correct_sum = 0;
  for (int i = 0; i < num_elements; i++) {
    correct_sum += h_in[i];
  }

  if (fabs(half_to_float(h_out) - correct_sum) > 0.001) {
    errors++;
    if (errors < 10) {
      printf("Expected %f, get h_out = %f\n", correct_sum,
             half_to_float(h_out));
    }
  }

  if (errors > 0) {
    printf("CUB_FULL_REDUCTION does not agree with SEQUENTIAL! %d errors!\n",
           errors);
  } else {
    printf("Results verified: they agree.\n\n");
  }

#endif

    hipFree(d_in_fp16);
    hipFree(d_out);
    hipFree(d_temp_storage);
  } catch (...) {
    hipFree(d_in_fp16);
    hipFree(d_out);
    hipFree(d_temp_storage);

    hipDeviceReset();
    const auto p = std::current_exception();
    std::rethrow_exception(p);
  }
}

BENCHMARK(CUB_FULL_REDUCTION)->ARGS()->UseManualTime();
