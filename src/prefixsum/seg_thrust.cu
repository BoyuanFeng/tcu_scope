#include "hip/hip_runtime.h"

#include <benchmark/benchmark.h>

#include "init/init.hpp"
#include "prefixsum/args.hpp"
#include "utils/utils.hpp"

#include <thrust/execution_policy.h>
#include <thrust/scan.h>

static void THRUST_SEGMENTED_PREFIXSUM(benchmark::State &state) {
  const size_t num_segments = state.range(0);
  const size_t segment_size = state.range(1);
  const size_t num_elements = num_segments * segment_size;

  int *h_keys = new int[num_elements];
  for (size_t i = 0; i < num_elements; i++) {
    h_keys[i] = i / segment_size;
  }

  int *d_keys     = nullptr;
  half *d_in_fp16 = nullptr;
  half *d_out     = nullptr;

  try {
    PRINT_IF_ERROR(hipMalloc(&d_keys, num_elements * sizeof(int)));
    PRINT_IF_ERROR(hipMalloc(&d_in_fp16, num_elements * sizeof(half)));
    PRINT_IF_ERROR(hipMalloc(&d_out, num_elements * sizeof(half)));

    PRINT_IF_ERROR(
        hipMemcpy(d_keys, h_keys, num_elements * sizeof(int), hipMemcpyHostToDevice));
    cuda_memory_set(d_in_fp16, 0.001f, num_elements);

    PRINT_IF_ERROR(hipDeviceSynchronize());

    hipEvent_t start, stop;
    PRINT_IF_ERROR(hipEventCreate(&start));
    PRINT_IF_ERROR(hipEventCreate(&stop));

    defer(hipEventDestroy(start));
    defer(hipEventDestroy(stop));

    for (auto _ : state) {
      PRINT_IF_ERROR(hipEventRecord(start));

      thrust::inclusive_scan_by_key(thrust::device, d_keys, d_keys + num_elements,
                                    d_in_fp16, d_out);

      PRINT_IF_ERROR(hipEventRecord(stop));
      PRINT_IF_ERROR(hipEventSynchronize(stop));

      state.PauseTiming();

      float msecTotal = 0.0f;
      PRINT_IF_ERROR(hipEventElapsedTime(&msecTotal, start, stop));
      state.SetIterationTime(msecTotal / 1000);
      state.ResumeTiming();
    }

    state.counters.insert({{"num_segments", num_segments},
                           {"num_elements", num_segments * segment_size},
                           {"segment_size", segment_size},
                           {"flops",
                            {state.iterations() * 1.0 * num_elements,
                             benchmark::Counter::kAvgThreadsRate}}});

#if 0
  half *h_out = new half[num_elements];
  PRINT_IF_ERROR(hipMemcpy(h_out, d_out, num_elements * sizeof(half), hipMemcpyDeviceToHost));

  int errors = 0;
  for (int j = 0; j < num_segments; j++) {
    float correct_segment_sum = 0;
    for (int i = 0; i < segment_size; i++) {
      correct_segment_sum += h_in[j * segment_size + i];
      if (fabs(half_to_float(h_out[j * segment_size + i]) - correct_segment_sum) > 0.001) {
        errors++;
        if (errors < 10) {
        printf("Expected %f, get h_out[%d] = %f\n", correct_segment_sum, i, half_to_float(h_out[j * segment_size + i]));
        }
        }
    }
  }

  if (errors > 0) {
    printf("CUB_SEGMENTED_PREFIXSUM does not agree with SEQUENTIAL! %d errors!\n", errors);
  } else {
    printf("Results verified: they agree.\n\n");
  }

  delete h_out;
#endif

    PRINT_IF_ERROR(hipFree(d_in_fp16));
    PRINT_IF_ERROR(hipFree(d_out));
    PRINT_IF_ERROR(hipFree(d_keys));
    delete h_keys;

  } catch (...) {
    hipFree(d_in_fp16);
    hipFree(d_out);
    hipFree(d_keys);
    delete h_keys;

    hipDeviceReset();
    const auto p = std::current_exception();
    std::rethrow_exception(p);
  }
}

BENCHMARK(THRUST_SEGMENTED_PREFIXSUM)->SEG_16_ARGS()->UseManualTime();
BENCHMARK(THRUST_SEGMENTED_PREFIXSUM)->SEG_32_ARGS()->UseManualTime();
BENCHMARK(THRUST_SEGMENTED_PREFIXSUM)->SEG_64_ARGS()->UseManualTime();
BENCHMARK(THRUST_SEGMENTED_PREFIXSUM)->SEG_128_ARGS()->UseManualTime();
BENCHMARK(THRUST_SEGMENTED_PREFIXSUM)->SEG_256_ARGS()->UseManualTime();
BENCHMARK(THRUST_SEGMENTED_PREFIXSUM)->SEG_512_ARGS()->UseManualTime();
BENCHMARK(THRUST_SEGMENTED_PREFIXSUM)->SEG_1024_ARGS()->UseManualTime();
BENCHMARK(THRUST_SEGMENTED_PREFIXSUM)->SEG_2048_ARGS()->UseManualTime();
BENCHMARK(THRUST_SEGMENTED_PREFIXSUM)->SEG_4096_ARGS()->UseManualTime();
BENCHMARK(THRUST_SEGMENTED_PREFIXSUM)->SEG_8192_ARGS()->UseManualTime();
BENCHMARK(THRUST_SEGMENTED_PREFIXSUM)->SEG_16384_ARGS()->UseManualTime();

static void THRUST_TUNE_SEGMENTED_PREFIXSUM(benchmark::State &state) {
  THRUST_SEGMENTED_PREFIXSUM(state);
}

#define RUN_THRUST_TUNE(TUNE_ARGS)                                                       \
  BENCHMARK(THRUST_TUNE_SEGMENTED_PREFIXSUM)->Apply(TUNE_ARGS)->UseManualTime();

// RUN_THRUST_TUNE(Tuning16_x_14);
// RUN_THRUST_TUNE(Tuning16_x_18);
RUN_THRUST_TUNE(Tuning16_x_22);
// RUN_THRUST_TUNE(Tuning16_x_26);
RUN_THRUST_TUNE(Tuning16_x_30);
