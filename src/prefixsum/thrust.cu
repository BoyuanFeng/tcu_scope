#include "hip/hip_runtime.h"

#include <benchmark/benchmark.h>

#include "init/init.hpp"
#include "prefixsum/args.hpp"
#include "utils/utils.hpp"

#include <thrust/execution_policy.h>
#include <thrust/scan.h>

static void THRUST_FULL_PREFIXSUM(benchmark::State &state) {
  const size_t num_elements = state.range(0);

  hipEvent_t start, stop;
  half *d_in_fp16 = nullptr;
  half *d_out     = nullptr;

  try {
    PRINT_IF_ERROR(hipMalloc(&d_in_fp16, num_elements * sizeof(half)));
    PRINT_IF_ERROR(hipMalloc(&d_out, num_elements * sizeof(half)));

    cuda_memory_set(d_in_fp16, 0.001f, num_elements);

    PRINT_IF_ERROR(hipDeviceSynchronize());

    PRINT_IF_ERROR(hipEventCreate(&start));
    PRINT_IF_ERROR(hipEventCreate(&stop));

    defer(hipEventDestroy(start));
    defer(hipEventDestroy(stop));

    for (auto _ : state) {
      PRINT_IF_ERROR(hipEventRecord(start));

      thrust::inclusive_scan(thrust::device, d_in_fp16, d_in_fp16 + num_elements, d_out);

      PRINT_IF_ERROR(hipEventRecord(stop));
      PRINT_IF_ERROR(hipEventSynchronize(stop));

      state.PauseTiming();

      float msecTotal = 0.0f;
      PRINT_IF_ERROR(hipEventElapsedTime(&msecTotal, start, stop));
      state.SetIterationTime(msecTotal / 1000);
      state.ResumeTiming();
    }

    state.counters.insert({{"num_elements", num_elements},
                           {"flops",
                            {state.iterations() * 1.0 * num_elements,
                             benchmark::Counter::kAvgThreadsRate}}});

#if 0
  half *h_out = new half[num_elements];
  PRINT_IF_ERROR(hipMemcpy(h_out, d_out, num_elements * sizeof(half), hipMemcpyDeviceToHost));

  int errors        = 0;
  float correct_sum = 0;
  for (int i = 0; i < num_elements; i++) {
    correct_sum += h_in[i];
    if (fabs(half_to_float((h_out[i])) - correct_sum) > 0.01) {
      errors++;
      printf("Expected %f, get h_out[%d] = %f\n", correct_sum, i, half_to_float(h_out[i]));
    }
  }

  if (errors > 0) {
    printf("THRUST_FULL_PREFIXSUM does not agree with SEQUENTIAL! %d errors!\n", errors);
  } else {
    printf("Results verified: they agree.\n\n");
  }

  delete h_out;
#endif

    hipFree(d_in_fp16);
  } catch (...) {
    hipFree(d_in_fp16);

    hipDeviceReset();
    const auto p = std::current_exception();
    std::rethrow_exception(p);
  }
}

BENCHMARK(THRUST_FULL_PREFIXSUM)->ARGS()->UseManualTime();
