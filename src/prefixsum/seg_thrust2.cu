#include "hip/hip_runtime.h"

#include <benchmark/benchmark.h>

#include "init/init.hpp"
#include "prefixsum/args.hpp"
#include "utils/utils.hpp"

#include <thrust/execution_policy.h>
#include <thrust/scan.h>

static void THRUST_SEGMENTED2_PREFIXSUM(benchmark::State &state) {
  const size_t num_segments = state.range(0);
  const size_t segment_size = state.range(1);
  const size_t num_elements = num_segments * segment_size;

  hipEvent_t start, stop;
  half *d_in_fp16 = nullptr;
  half *d_out     = nullptr;

  try {
    PRINT_IF_ERROR(hipMalloc(&d_in_fp16, num_elements * sizeof(half)));
    PRINT_IF_ERROR(hipMalloc(&d_out, num_elements * sizeof(half)));

    cuda_memory_set(d_in_fp16, 0.001f, num_elements);

    PRINT_IF_ERROR(hipDeviceSynchronize());

    PRINT_IF_ERROR(hipEventCreate(&start));
    PRINT_IF_ERROR(hipEventCreate(&stop));

    defer(hipEventDestroy(start));
    defer(hipEventDestroy(stop));

    for (auto _ : state) {
      PRINT_IF_ERROR(hipEventRecord(start));

      for (size_t ii = 0; ii < num_segments; ii++) {
        thrust::inclusive_scan(thrust::device, d_in_fp16 + ii * segment_size,
                               d_in_fp16 + (ii + 1) * segment_size,
                               d_out + ii * segment_size);
      }

      PRINT_IF_ERROR(hipEventRecord(stop));
      PRINT_IF_ERROR(hipEventSynchronize(stop));

      state.PauseTiming();

      float msecTotal = 0.0f;
      PRINT_IF_ERROR(hipEventElapsedTime(&msecTotal, start, stop));
      state.SetIterationTime(msecTotal / 1000);
      state.ResumeTiming();
    }

    state.counters.insert({{"num_segments", num_segments},
                           {"num_elements", num_segments * segment_size},
                           {"segment_size", segment_size},
                           {"flops",
                            {state.iterations() * 1.0 * num_elements,
                             benchmark::Counter::kAvgThreadsRate}}});

#if 0
  half *h_out = new half[num_elements];
  PRINT_IF_ERROR(hipMemcpy(h_out, d_out, num_elements * sizeof(half), hipMemcpyDeviceToHost));

  int errors        = 0;
  float correct_sum = 0;
  for (int i = 0; i < num_elements; i++) {
    correct_sum += h_in[i];
    if (fabs(half_to_float((h_out[i])) - correct_sum) > 0.01) {
      errors++;
      printf("Expected %f, get h_out[%d] = %f\n", correct_sum, i, half_to_float(h_out[i]));
    }
  }

  if (errors > 0) {
    printf("THRUST_SEGMENTED2_PREFIXSUM does not agree with SEQUENTIAL! %d errors!\n", errors);
  } else {
    printf("Results verified: they agree.\n\n");
  }

  delete h_out;
#endif

    hipFree(d_in_fp16);
  } catch (...) {
    hipFree(d_in_fp16);

    hipDeviceReset();
    const auto p = std::current_exception();
    std::rethrow_exception(p);
  }
}

BENCHMARK(THRUST_SEGMENTED2_PREFIXSUM)->SEG_16_ARGS()->UseManualTime();
BENCHMARK(THRUST_SEGMENTED2_PREFIXSUM)->SEG_32_ARGS()->UseManualTime();
BENCHMARK(THRUST_SEGMENTED2_PREFIXSUM)->SEG_64_ARGS()->UseManualTime();
BENCHMARK(THRUST_SEGMENTED2_PREFIXSUM)->SEG_128_ARGS()->UseManualTime();
BENCHMARK(THRUST_SEGMENTED2_PREFIXSUM)->SEG_256_ARGS()->UseManualTime();
BENCHMARK(THRUST_SEGMENTED2_PREFIXSUM)->SEG_512_ARGS()->UseManualTime();
BENCHMARK(THRUST_SEGMENTED2_PREFIXSUM)->SEG_1024_ARGS()->UseManualTime();
BENCHMARK(THRUST_SEGMENTED2_PREFIXSUM)->SEG_2048_ARGS()->UseManualTime();
BENCHMARK(THRUST_SEGMENTED2_PREFIXSUM)->SEG_4096_ARGS()->UseManualTime();
BENCHMARK(THRUST_SEGMENTED2_PREFIXSUM)->SEG_8192_ARGS()->UseManualTime();
BENCHMARK(THRUST_SEGMENTED2_PREFIXSUM)->SEG_16384_ARGS()->UseManualTime();

static void THRUST_TUNE_SEGMENTED2_PREFIXSUM(benchmark::State &state) {
  THRUST_SEGMENTED2_PREFIXSUM(state);
}

#define RUN_THRUST_TUNE(TUNE_ARGS)                                                       \
  BENCHMARK(THRUST_TUNE_SEGMENTED2_PREFIXSUM)->Apply(TUNE_ARGS)->UseManualTime();

// RUN_THRUST_TUNE(Tuning16_x_14);
// RUN_THRUST_TUNE(Tuning16_x_18);
RUN_THRUST_TUNE(Tuning16_x_22);
// RUN_THRUST_TUNE(Tuning16_x_26);
RUN_THRUST_TUNE(Tuning16_x_30);
