#include "hip/hip_runtime.h"

#include <benchmark/benchmark.h>

#include "init/init.hpp"
#include "unsafe_reduction/args.hpp"
#include "utils/utils.hpp"

#include "kernel.cuh"

using namespace wmma_unsafe_reduction;

template <size_t SEGMENT_SIZE, int WARPS_PER_BLOCK>
static void tryCUDA_UNSAFE_WMMA_SEGMENTED_REDUCTION_16N_BLOCK(benchmark::State &state) {
  const size_t num_segments = state.range(0);
  const size_t segment_size = state.range(1);

  if (segment_size != SEGMENT_SIZE) {
    state.SkipWithError(fmt::format("segment_size={} must be equal to SEGMENT_SIZE={} ",
                                    segment_size, SEGMENT_SIZE)
                            .c_str());
    return;
  }

  const int BLOCK_DIM       = WARPS_PER_BLOCK * WARP_SIZE;
  const size_t num_elements = num_segments * segment_size;

  defer(hipDeviceReset());

  half *d_in_fp16 = nullptr;
  half *d_out     = nullptr;
  hipEvent_t start, stop;

  try {
    PRINT_IF_ERROR(hipMalloc(&d_in_fp16, num_elements * sizeof(half)));
    PRINT_IF_ERROR(hipMalloc(&d_out, num_segments * sizeof(half)));

    cuda_memory_set(d_in_fp16, 0.001f, num_elements);

    dim3 gridDim, blockDim;
    blockDim.x = BLOCK_DIM;
    gridDim.x  = num_segments / 16;

    if (gridDim.x >= CUDA_MAX_GRID_SIZE) {
      state.SkipWithError(
          fmt::format("gridDim.x={} is greater than CUDA_MAX_GRID_SIZE", gridDim.x)
              .c_str());
      return;
    }

    if (gridDim.x == 0) {
      state.SkipWithError(fmt::format("gridDim.x={} is 0", gridDim.x).c_str());
      return;
    }

    PRINT_IF_ERROR(hipEventCreate(&start));
    PRINT_IF_ERROR(hipEventCreate(&stop));

    defer(hipEventDestroy(start));
    defer(hipEventDestroy(stop));

    for (auto _ : state) {
      PRINT_IF_ERROR(hipEventRecord(start));

      compute_wmma_segmented_reduction_16n_block<SEGMENT_SIZE, WARPS_PER_BLOCK, BLOCK_DIM>
          <<<gridDim, blockDim>>>(d_in_fp16, d_out, num_segments);

      PRINT_IF_ERROR(hipEventRecord(stop));
      PRINT_IF_ERROR(hipEventSynchronize(stop));

      /* state.SkipWithError("break"); */
      state.PauseTiming();

      float msecTotal = 0.0f;
      PRINT_IF_ERROR(hipEventElapsedTime(&msecTotal, start, stop));
      state.SetIterationTime(msecTotal / 1000);
      state.ResumeTiming();
    }

    state.counters.insert({{"num_segments", num_segments},
                           {"segment_size", segment_size},
                           {"num_elements", num_segments * segment_size},
                           {"warps_per_block", WARPS_PER_BLOCK},
                           {"flops",
                            {state.iterations() * 1.0 * num_segments * segment_size,
                             benchmark::Counter::kAvgThreadsRate}}});
#if 0
  half *h_out = new half[num_segments];
  PRINT_IF_ERROR(hipMemcpy(h_out, d_out, num_segments * sizeof(half),
                            hipMemcpyDeviceToHost));

  int errors = 0;
  for (int j = 0; j < num_segments; j++) {
    float correct_segment_sum = 0;
    for (int i = 0; i < segment_size; i++) {
      correct_segment_sum += h_in[j * segment_size + i];
    }
    if (fabs(half_to_float(h_out[j]) - correct_segment_sum) > 0.1) {
      errors++;
      printf("Expected %f, get h_out[%d] = %f\n", correct_segment_sum, j,
             half_to_float(h_out[j]));
    }
  }

  if (errors > 0) {
    printf(
        "CUDA_UNSAFE_WMMA_SEGMENTED_REDUCTION_16N_BLOCK does not agree with SEQUENTIAL! %d errors!\n",
        errors);
  } else {
    printf("Results verified: they agree.\n\n");
  }

  delete h_out;
#endif

    hipFree(d_in_fp16);
    hipFree(d_out);
  } catch (...) {
    hipFree(d_in_fp16);
    hipFree(d_out);

    hipDeviceReset();
    const auto p = std::current_exception();
    std::rethrow_exception(p);
  }
}

template <size_t SEGMENT_SIZE, int WARPS_PER_BLOCK>
static void iCUDA_UNSAFE_WMMA_SEGMENTED_REDUCTION_16N_BLOCK(benchmark::State &state) {
  hipDeviceReset();
  try {
    tryCUDA_UNSAFE_WMMA_SEGMENTED_REDUCTION_16N_BLOCK<SEGMENT_SIZE, WARPS_PER_BLOCK>(
        state);
  } catch (const std::exception &e) {
    state.SkipWithError(e.what());
  } catch (const std::string &e) {
    state.SkipWithError(e.c_str());
  } catch (...) {
    state.SkipWithError("unknown exception");
  }
}

template <int WARPS_PER_BLOCK>
static void CUDA_UNSAFE_WMMA_SEGMENTED_REDUCTION_16N_BLOCK(benchmark::State &state) {
  const int segment_size = state.range(1);
  switch (segment_size) {
#define Dispatch(N)                                                                      \
  case N:                                                                                \
    iCUDA_UNSAFE_WMMA_SEGMENTED_REDUCTION_16N_BLOCK<N, WARPS_PER_BLOCK>(state);          \
    break

    Dispatch(16);
    Dispatch(32);
    Dispatch(64);
    Dispatch(128);
    Dispatch(256);
    Dispatch(512);
    Dispatch(1024);
    Dispatch(2048);
    Dispatch(4096);
    Dispatch(8192);
    Dispatch(16384);
    Dispatch(32768);
    Dispatch(65536);
    Dispatch(131072);
    Dispatch(262144);
    Dispatch(524288);
    Dispatch(1048576);
    Dispatch(2097152);
    Dispatch(4194304);
    Dispatch(8388608);
    Dispatch(16777216);
    Dispatch(33554432);
    Dispatch(67108864);
    Dispatch(134217728);
    Dispatch(268435456);
    Dispatch(536870912);
    Dispatch(1073741824);
    default:
      static_assert(true, "invalid segment size");
      state.SkipWithError("invalid segment size");
#undef DISPATCH
  }
}

template <int WARPS_PER_BLOCK>
static void CUDA_UNSAFE_WMMA_TUNE_SEGMENTED_REDUCTION_16N_BLOCK(benchmark::State &state) {
  CUDA_UNSAFE_WMMA_SEGMENTED_REDUCTION_16N_BLOCK<WARPS_PER_BLOCK>(state);
}

#define RUN_CUDA_UNSAFE_WMMA_TUNE(TUNE_ARGS)                                             \
  BENCHMARK_TEMPLATE(CUDA_UNSAFE_WMMA_TUNE_SEGMENTED_REDUCTION_16N_BLOCK, 1)             \
      ->Apply(TUNE_ARGS)                                                                 \
      ->UseManualTime();                                                                 \
  BENCHMARK_TEMPLATE(CUDA_UNSAFE_WMMA_TUNE_SEGMENTED_REDUCTION_16N_BLOCK, 2)             \
      ->Apply(TUNE_ARGS)                                                                 \
      ->UseManualTime();                                                                 \
  BENCHMARK_TEMPLATE(CUDA_UNSAFE_WMMA_TUNE_SEGMENTED_REDUCTION_16N_BLOCK, 4)             \
      ->Apply(TUNE_ARGS)                                                                 \
      ->UseManualTime();                                                                 \
  BENCHMARK_TEMPLATE(CUDA_UNSAFE_WMMA_TUNE_SEGMENTED_REDUCTION_16N_BLOCK, 8)             \
      ->Apply(TUNE_ARGS)                                                                 \
      ->UseManualTime();                                                                 \
  BENCHMARK_TEMPLATE(CUDA_UNSAFE_WMMA_TUNE_SEGMENTED_REDUCTION_16N_BLOCK, 16)            \
      ->Apply(TUNE_ARGS)                                                                 \
      ->UseManualTime();

// RUN_CUDA_UNSAFE_WMMA_TUNE(Tuning16_x_14);
// RUN_CUDA_UNSAFE_WMMA_TUNE(Tuning16_x_18);
// RUN_CUDA_UNSAFE_WMMA_TUNE(Tuning16_x_22);
// RUN_CUDA_UNSAFE_WMMA_TUNE(Tuning16_x_26);
RUN_CUDA_UNSAFE_WMMA_TUNE(Tuning16_x_30);

#define RUN_CUDA_UNSAFE_WMMA(Args)                                                       \
  BENCHMARK_TEMPLATE(CUDA_UNSAFE_WMMA_SEGMENTED_REDUCTION_16N_BLOCK, 1)                  \
      ->Args()                                                                           \
      ->UseManualTime();                                                                 \
  BENCHMARK_TEMPLATE(CUDA_UNSAFE_WMMA_SEGMENTED_REDUCTION_16N_BLOCK, 2)                  \
      ->Args()                                                                           \
      ->UseManualTime();                                                                 \
  BENCHMARK_TEMPLATE(CUDA_UNSAFE_WMMA_SEGMENTED_REDUCTION_16N_BLOCK, 4)                  \
      ->Args()                                                                           \
      ->UseManualTime();                                                                 \
  BENCHMARK_TEMPLATE(CUDA_UNSAFE_WMMA_SEGMENTED_REDUCTION_16N_BLOCK, 8)                  \
      ->Args()                                                                           \
      ->UseManualTime();                                                                 \
  BENCHMARK_TEMPLATE(CUDA_UNSAFE_WMMA_SEGMENTED_REDUCTION_16N_BLOCK, 16)                 \
      ->Args()                                                                           \
      ->UseManualTime();

RUN_CUDA_UNSAFE_WMMA(SEG_16_ARGS);
RUN_CUDA_UNSAFE_WMMA(SEG_32_ARGS);
RUN_CUDA_UNSAFE_WMMA(SEG_64_ARGS);
RUN_CUDA_UNSAFE_WMMA(SEG_128_ARGS);
RUN_CUDA_UNSAFE_WMMA(SEG_256_ARGS);
RUN_CUDA_UNSAFE_WMMA(SEG_512_ARGS);
RUN_CUDA_UNSAFE_WMMA(SEG_1024_ARGS);
